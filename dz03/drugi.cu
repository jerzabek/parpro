
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 256

/**
 * Ubrzanje slijednog algoritma u usporedbi sa paralelnim algoritmom

  N	          SEQ	      Parallel	Ubrzanje
  10000	      1,9	      0,4	      4,75
  100000	    17,7	    1,6	      11,0625
  1000000	    175,8	    13,2	    13,31818182
  10000000	  1357,4	  127,7	    10,62960063

*/

__global__ void setup_kernel(hiprandState *state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  hiprand_init(1337, i, i, &state[i]);
}

__global__ void generate_kernel(hiprandState *state, unsigned int *counters) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  float x = hiprand_uniform(&state[i]);
  float y = hiprand_uniform(&state[i]);

  if (x * x + y * y <= 1.0f) atomicAdd(counters, 1);
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Missing argument: %s <number of points>\n", argv[0]);
    return 1;
  }

  int N = atoi(argv[1]);

  hiprandState *devStates;
  unsigned int *devCounters;
  unsigned int hostCounters = 0;

  hipEvent_t start, stop;
  float time;

  hipMalloc((void **)&devCounters, sizeof(unsigned int));
  hipMemset(devCounters, 0, sizeof(unsigned int));

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  hipMalloc((void **)&devStates, THREADS_PER_BLOCK * sizeof(hiprandState));

  setup_kernel<<<1, THREADS_PER_BLOCK>>>(devStates);

  for (int i = 0; i < N; i += THREADS_PER_BLOCK) {
    generate_kernel<<<1, THREADS_PER_BLOCK>>>(devStates, devCounters);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
      printf("Error: %s\n", hipGetErrorString(err));
  }

  hipMemcpy(&hostCounters, devCounters, sizeof(unsigned int), hipMemcpyDeviceToHost);

  printf("Pi is basically = %f\n", 4.0 * hostCounters / N);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Time:  %3.1f ms \n", time);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(devStates);
  hipFree(devCounters);

  return 0;
}
